#include "hip/hip_runtime.h"
#include "linear.h"
#include "../utils/utils.h"


// A "linear layer" neural network performs a linear transformation on its input: Y = xw^t + b
// this is a neural network that performs a linear transformation on its input
/*
 *X: input matrix (batch_size, input_features)
 *W: weights matrix (output_features, input_features)
 *b: Bias vector (output_features)
 *Y: Output matrix (batch_size, output_features)
 *
 *The GPU implementation aims to parallelize these matrix operations operations
 */

__global__
void linear_forward_gpu(float *inp, float *weights, float *bias, float *out, int bs, int n_in, int n_out){ // Chain together the series of modules passed to the constructor
    int row = blockDim.x*blockIdx.x + threadIdx.x, col = blockDim.y*blockIdx.y + threadIdx.y;
    int ind_inp, ind_weights, ind_out;

    if ((row < bs) && (col < n_out)){
        ind_out = row*n_out + col;
        out[ind_out] = bias[col];

        for (int i=0; i<n_in; i++){
            ind_inp = row*n_in + i;
            ind_weights = i*n_out + col;

            out[ind_out] += inp[ind_inp]*weights[ind_weights];
        }
    }
}


__global__
void linear_backward_gpu(float *inp, float *weights, float *out, int bs, int n_in, int n_out){
    int row = blockDim.x*blockIdx.x + threadIdx.x, col = blockDim.y*blockIdx.y + threadIdx.y;
    int ind_inp, ind_weights, ind_out;

    if ((row < bs) && (col < n_out)){
        ind_out = row*n_out + col;

        for (int i=0; i<n_in; i++){
            ind_inp = row*n_in + i;
            ind_weights = i*n_out + col;

            atomicAdd(&inp[ind_inp], weights[ind_weights]*out[ind_out]);
        }
    }
}


__global__
void linear_update_gpu(float *inp, float *weights, float *bias, float *out, int bs, int n_in, int n_out, float lr){
    int row = blockDim.x*blockIdx.x + threadIdx.x, col = blockDim.y*blockIdx.y + threadIdx.y;
    int ind_inp, ind_weights, ind_out;

    if ((row < bs) && (col < n_out)){
        ind_out = row*n_out + col;
        atomicAdd(&bias[col], -lr*out[ind_out]);

        for (int i=0; i<n_in; i++){
            ind_inp = row*n_in + i;
            ind_weights = i*n_out + col;

            atomicAdd(&weights[ind_weights], -lr*inp[ind_inp]*out[ind_out]);
        }
    }
}


Linear_GPU::Linear_GPU(int _bs, int _n_in, int _n_out, float _lr){
    bs = _bs;
    n_in = _n_in;
    n_out = _n_out;
    lr = _lr;

    sz_weights = n_in*n_out;
    sz_out = bs*n_out;
    n_block_rows = (bs + block_size - 1) / block_size;
    n_block_cols = (n_out + block_size - 1) / block_size;

    hipMallocManaged(&weights, sz_weights*sizeof(float));
    hipMallocManaged(&bias, n_out*sizeof(float));

    kaiming_init(weights, n_in, n_out);
    init_zero(bias, n_out);
}


void Linear_GPU::forward(float *_inp, float *_out){
    inp = _inp;
    out = _out;

    dim3 n_blocks(n_block_rows, n_block_cols);
    dim3 n_threads(block_size, block_size);

    linear_forward_gpu<<<n_blocks, n_threads>>>(inp, weights, bias, out, bs, n_in, n_out);
    hipDeviceSynchronize();
}


void Linear_GPU::backward(){
    init_zero(inp, bs*n_in);

    dim3 n_blocks(n_block_rows, n_block_cols);
    dim3 n_threads(block_size, block_size);

    linear_backward_gpu<<<n_blocks, n_threads>>>(inp, cp_weights, out, bs, n_in, n_out);
    hipDeviceSynchronize();

    hipFree(cp_weights);
    hipFree(out);
}


void Linear_GPU::update(){
    hipMallocManaged(&cp_weights, sz_weights*sizeof(float));
    set_eq(cp_weights, weights, sz_weights);

    dim3 n_blocks(n_block_rows, n_block_cols);
    dim3 n_threads(block_size, block_size);

    linear_update_gpu<<<n_blocks, n_threads>>>(inp, weights, bias, out, bs, n_in, n_out, lr);
    hipDeviceSynchronize();
}
